
#include <hip/hip_runtime.h>
extern "C"
__global__ void initDbIndexKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predBaseIdx, int *d_predVarMat, int *d_dbIndex,
                                  long totalGroundings)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings)
    {
      long baseDbIndex = idx * totalPreds;
      for(int i = 0; i < totalPreds; i++)
	  d_dbIndex[baseDbIndex + i] = d_predBaseIdx[i];

      long n = idx;
      for(int i = totalVars-1; i >= 0; i--)
	{
	  int domainSize = d_varDomainSizes[i];
	  long temp = n / domainSize;
	  int val = n - temp * domainSize;
	  n = temp;

	  int basePredVarMatIndex = i * totalPreds;
	  for(int j = 0; j < totalPreds; j++)
	      d_dbIndex[baseDbIndex + j] += d_predVarMat[basePredVarMatIndex + j] * val;
	}
    }
}

extern "C"
__global__ void evalClauseKernel(int *d_satArray, int **d_interpretation, int *dbIndex,
                                 int *d_predicates, int *d_valTrue, int totalPreds, long totalGroundings)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
      long baseDbIndex = idx * totalPreds;
      int sat = 0;
      for(int i = 0; i < totalPreds; i++)
	{
	  int predId = d_predicates[i];
	  long interpretationIdx = dbIndex[baseDbIndex + i];
	  sat = max(sat, d_interpretation[predId][interpretationIdx] == d_valTrue[i]);
	}

      d_satArray[idx] = sat;
    }
}

extern "C"
__global__ void evalClauseWithoutDbKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predicates, int *d_predBaseIdx, int *d_valTrue, int *d_predVarMat, 
								  int *d_satArray, int **d_interpretation, long totalGroundings, int *d_mem)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
		int memBase = idx * totalVars;
		long n = idx;
		for(int i = totalVars-1; i >= 0; i--)
		{
		  int domainSize = d_varDomainSizes[i];
		  long temp = n / domainSize;
		  int val = n - temp * domainSize;
		  n = temp;
		  d_mem[memBase + i] = val;
		}

		int sat = 0;
		for(int i = 0; i < totalPreds; i++)
		{
			int predId = d_predicates[i];
			int dbIndex = d_predBaseIdx[i];
			for(int j = 0; j < totalVars; j++)
				dbIndex += d_mem[memBase + j] * d_predVarMat[j * totalPreds + i];
			sat = max(sat, d_interpretation[predId][dbIndex] == d_valTrue[i]);
		}
		d_satArray[idx] = sat;
    }
}

/*extern "C"
__global__ void evalClauseWithoutDbKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predicates, int *d_predBaseIdx, int *d_valTrue, int *d_predVarMat, 
								  int *d_satArray, int **d_interpretation, long totalGroundings)
{
  int dbIndex[5];

  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
		//long baseDbIndex = idx * totalPreds;
		for(int i = 0; i < totalPreds; i++)
			dbIndex[i] = d_predBaseIdx[i];
			//d_dbIndex[baseDbIndex + i] = d_predBaseIdx[i];

		long n = idx;
		for(int i = totalVars-1; i >= 0; i--)
		{
		  int domainSize = d_varDomainSizes[i];
		  long temp = n / domainSize;
		  int val = n - temp * domainSize;
		  n = temp;

		  int basePredVarMatIndex = i * totalPreds;
		  for(int j = 0; j < totalPreds; j++)
			dbIndex[j] += d_predVarMat[basePredVarMatIndex + j] * val;
			  //d_dbIndex[baseDbIndex + j] += d_predVarMat[basePredVarMatIndex + j] * val;
		}

		int sat = 0;
		for(int i = 0; i < totalPreds; i++)
		{
			int predId = d_predicates[i];
			long interpretationIdx = dbIndex[i];
			sat = max(sat, d_interpretation[predId][interpretationIdx] == d_valTrue[i]);
		}
		d_satArray[idx] = sat;
    }
}*/