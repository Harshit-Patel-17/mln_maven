
#include <hip/hip_runtime.h>
extern "C"
__global__ void initDbIndexKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predBaseIdx, int *d_predVarMat, int *d_dbIndex,
                                  long totalGroundings)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings)
    {
      long baseDbIndex = idx * totalPreds;
      for(int i = 0; i < totalPreds; i++)
	  d_dbIndex[baseDbIndex + i] = d_predBaseIdx[i];

      long n = idx;
      for(int i = totalVars-1; i >= 0; i--)
	{
	  int domainSize = d_varDomainSizes[i];
	  long temp = n / domainSize;
	  int val = n - temp * domainSize;
	  n = temp;

	  int basePredVarMatIndex = i * totalPreds;
	  for(int j = 0; j < totalPreds; j++)
	      d_dbIndex[baseDbIndex + j] += d_predVarMat[basePredVarMatIndex + j] * val;
	}
    }
}

extern "C"
__global__ void evalClauseKernel(int *d_satArray, int **d_interpretation, int *dbIndex,
                                 int *d_predicates, int *d_valTrue, int totalPreds, long totalGroundings)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
      long baseDbIndex = idx * totalPreds;
      int sat = 0;
      for(int i = 0; i < totalPreds; i++)
	{
	  int predId = d_predicates[i];
	  long interpretationIdx = dbIndex[baseDbIndex + i];
	  sat = max(sat, d_interpretation[predId][interpretationIdx] == d_valTrue[i]);
	}

      d_satArray[idx] = sat;
    }
}

extern "C"
__global__ void evalClauseWithoutDbKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predicates, int *d_negated, int *d_predBaseIdx, int *d_valTrue, int *d_predVarMat, 
								  int *d_satArray, int **d_interpretation, long totalGroundings, long offset, int *d_mem)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
		int memBase = idx * totalVars;
		long n = idx + offset;
		for(int i = totalVars-1; i >= 0; i--)
		{
		  int domainSize = d_varDomainSizes[i];
		  long temp = n / domainSize;
		  int val = n - temp * domainSize;
		  n = temp;
		  d_mem[memBase + i] = val;
		}

		int sat = 0;
		for(int i = 0; i < totalPreds; i++)
		{
			int predId = d_predicates[i];
			int negated = d_negated[i];
			int dbIndex = d_predBaseIdx[i];
			for(int j = 0; j < totalVars; j++)
				dbIndex += d_mem[memBase + j] * d_predVarMat[j * totalPreds + i];
			if(negated == 0)
				sat = max(sat, d_interpretation[predId][dbIndex] == d_valTrue[i]);
			else
				sat = max(sat, d_interpretation[predId][dbIndex] != d_valTrue[i]);
		}
		d_satArray[idx] = sat;
    }
}

extern "C"
__global__ void evalCNFKernel(int totalVars, int totalClauses, int *totalPredsInClause, int *d_varDomainSizes,
                              int **d_predicates, int **d_negated, int **d_predBaseIdx, int **d_valTrue,
                              int **d_predVarMat, int *d_satArray, int **d_interpretation, long totalGroundings,
                              long offset, int *d_mem)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings)
    {
		int memBase = idx * totalVars;
		long n = idx + offset;
		for(int i = totalVars-1; i >= 0; i--)
		{
		  int domainSize = d_varDomainSizes[i];
		  long temp = n / domainSize;
		  int val = n - temp * domainSize;
		  n = temp;
		  d_mem[memBase + i] = val;
		}

		int sat = 1;
		for(int c = 0; c < totalClauses; c++) {
		  if(sat == 0)
		    break;
		  int clauseSat = 0;
		  int totalPredicates = totalPredsInClause[c];
		  for(int i = 0; i < totalPredicates; i++)
		  {
			  int predId = d_predicates[c][i];
			  int negated = d_negated[c][i];
			  int dbIndex = d_predBaseIdx[c][i];
			  for(int j = 0; j < totalVars; j++)
				  dbIndex += d_mem[memBase + j] * d_predVarMat[c][j * totalPredicates + i];
			  if(negated == 0)
			    clauseSat = max(clauseSat, d_interpretation[predId][dbIndex] == d_valTrue[c][i]);
			  else
			    clauseSat = max(clauseSat, d_interpretation[predId][dbIndex] != d_valTrue[c][i]);
		  }
		  sat = min(sat, clauseSat);
		}
		d_satArray[idx] = sat;
    }
}

/*extern "C"
__global__ void evalClauseWithoutDbKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predicates, int *d_predBaseIdx, int *d_valTrue, int *d_predVarMat, 
								  int *d_satArray, int **d_interpretation, long totalGroundings)
{
  int dbIndex[5];

  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
		//long baseDbIndex = idx * totalPreds;
		for(int i = 0; i < totalPreds; i++)
			dbIndex[i] = d_predBaseIdx[i];
			//d_dbIndex[baseDbIndex + i] = d_predBaseIdx[i];

		long n = idx;
		for(int i = totalVars-1; i >= 0; i--)
		{
		  int domainSize = d_varDomainSizes[i];
		  long temp = n / domainSize;
		  int val = n - temp * domainSize;
		  n = temp;

		  int basePredVarMatIndex = i * totalPreds;
		  for(int j = 0; j < totalPreds; j++)
			dbIndex[j] += d_predVarMat[basePredVarMatIndex + j] * val;
			  //d_dbIndex[baseDbIndex + j] += d_predVarMat[basePredVarMatIndex + j] * val;
		}

		int sat = 0;
		for(int i = 0; i < totalPreds; i++)
		{
			int predId = d_predicates[i];
			long interpretationIdx = dbIndex[i];
			sat = max(sat, d_interpretation[predId][interpretationIdx] == d_valTrue[i]);
		}
		d_satArray[idx] = sat;
    }
}*/
